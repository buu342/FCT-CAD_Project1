#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <assert.h>
#include <time.h>
#include <ctype.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>



/*********************************
              Macros
*********************************/

// Print the image before and after to the console
#define PRINT_PPM    0

// The radius of the blur matrix kernel
#define KERNELRADIUS 2

// Useful constants. Don't touch
#define KERNELSIZE   ((2*KERNELRADIUS)-1)
#define TWO_PI 6.28319
#define EULER 2.71828


/*********************************
            Structures
*********************************/

// Texel struct
typedef uint8_t color;
typedef struct {
    color r;
    color g;
    color b;
    color alignment; // Unused, exists for 32 bit memory alignment.
} texel;

// Blur matrix struct (workaround for passing arrays by value)
typedef struct {
  float data[KERNELSIZE];
} blurMatrix;


/*==============================
    read_ppm
    Reads a PPM image ascii file
    @uint32_tparam The pointer to the image file handler 
    @param The pointer to an array of texels
    @param A pointer to store the image width 
    @param A pointer to store the image height
    @param A pointer to store the size of each texel 
==============================*/

void read_ppm(FILE* f, texel** img, uint32_t* width, uint32_t* height, uint32_t* texelsize)
{
    color r, g, b;
    int c;
    uint32_t count = 0;
    char ppm[10];

    // Read the header to get the image properties
    while ((c = fgetc(f)) != EOF && count < 4)
    {

        // Ignore whitespace
        if (isspace(c)) 
            continue;

        // Ignore comments
        if (c == '#') 
        {
            while (fgetc(f) != '\n')
                ;
            continue;
        }

        // Read header elements
        ungetc(c, f);
        switch (count) 
        {
            case 0: count += fscanf(f, "%2s", ppm); break;
            case 1: count += fscanf(f, "%u%u%u", width, height, texelsize); break;
            case 2: count += fscanf(f, "%u%u", height, texelsize); break;
            case 3: count += fscanf(f, "%u", texelsize);
        }
    }

    // Validate what we read
    assert(c != EOF);
    assert(!strcmp("P3", ppm));

    // Allocate memory for the image data
    *img = (texel*)malloc((*width)*(*height)*sizeof(texel));
    assert(img != NULL);

    // Read the texel data from the file
    int pos = 0;
    while (fscanf(f,"%hhu%hhu%hhu", &r, &g, &b) == 3)
    {
        texel* t = &(*img)[pos];
        t->r = r;
        t->g = g;
        t->b = b;
        pos++;
    }
    assert(pos == (*width)*(*height));
}


/*==============================
    write_ppm
    Writes a PPM image ascii file
    @param The pointer to the image file handler 
    @param The image data
    @param The image width (in texels)
    @param The image height (in texels)
    @param The size of each texel 
==============================*/

void write_ppm(FILE* f, texel* img, const uint32_t width, const uint32_t height, const uint32_t texelsize)
{
    // Write the header
    fprintf(f, "P3\n%u %u %u\n", width, height, texelsize);

    // Write the texel data
    for (uint32_t l=0; l<height; l++)
    {
        for (uint32_t c=0; c<width; c++) 
        {
            uint32_t p = (l*width + c);
            fprintf(f, "%hhu %hhu %hhu  ", img[p].r, img[p].g, img[p].b);
        }
        putc('\n', f);
    }
}


/*==============================
    write_ppm
    Prints a PPM image to the screen
    @param The image width (in texels)
    @param The image height (in texels)
    @param The image data
==============================*/

void printImg(const uint32_t imgw, const uint32_t imgh, const texel* img)
{
    for (uint32_t j=0; j<imgh; j++) 
    {
        for (uint32_t i=0; i<imgw; i++) 
        {
            uint32_t x = (i+j*imgw);
            printf("%hhu,%hhu,%hhu  ", img[x].r, img[x].g, img[x].b);
        }
        putchar('\n');
    }
    putchar('\n');
}


/*==============================
    shader_pass1
    Performs a horizontal gaussian blur with CUDA
    @param The image array to output to
    @param The image array to read from
    @param The size of the image (in texels)
    @param The image width (in texels)
    @param The image height (in texels)
    @param The gaussian blur matrix
    @param The amount of desaturation (ranging from 0 to 1).
==============================*/

__global__ void shader_pass1(texel* out, texel* in, const uint32_t size, const uint32_t width, const uint32_t height, const blurMatrix filter, const uint32_t sharedmemCount)
{
    extern __shared__ int s[];

    // Calculate the texel coordinate that this thread will modify
    const uint32_t index = blockIdx.x*blockDim.x + threadIdx.x;
    const uint32_t sindex = (KERNELRADIUS-1) + index%sharedmemCount;
    texel* sharedTexel = (texel*)&s;

    // Ensure we don't go out of bounds
    if (index >= size)
        return;

    // Calculate the texel coordinates
    float r=0, g=0, b=0, n=0;
    const int idx = index%width;
    const uint32_t upperx = sindex+(KERNELRADIUS-1);

    // Copy shared memory
    sharedTexel[sindex] = in[index];
    if (sindex == (KERNELRADIUS-1))
        for (int i=min(idx, KERNELRADIUS)-1; i>0; i--)
            sharedTexel[sindex-i] = in[index-i];
    if (sindex+1 == sharedmemCount)
        for (int i=min(width-idx, KERNELRADIUS)-1; i>0; i--)
            sharedTexel[sindex+i] = in[index+i];
    __syncthreads();

    // Perform a first horizontal gaussian blur pass
    for (int x=sindex-(KERNELRADIUS-1), k=-min(0, idx-(KERNELRADIUS-1)); x<upperx; x++)
    {
        const float scale = filter.data[k++];
        const texel* ti = &sharedTexel[x];
        r += scale*ti->r;
        g += scale*ti->g;
        b += scale*ti->b;
        n += scale;
    }
    n = 1/n;

    // Store this first pass
    texel* to = &out[index];
    to->r = r*n;
    to->g = g*n;
    to->b = b*n;
}


/*==============================
    shader_pass2
    Performs a vertical gaussian blur, and then desaturation with CUDA
    @param The image array to output to
    @param The image array to read from
    @param The size of the image (in texels)
    @param The image width (in texels)
    @param The image height (in texels)
    @param The gaussian blur matrix
    @param The amount of desaturation (ranging from 0 to 1).
==============================*/

__global__ void shader_pass2(texel* img, const uint32_t size, const uint32_t width, const uint32_t height, const blurMatrix filter, const float desaturation, const uint32_t sharedmemCount)
{
    extern __shared__ int s[];

    // Calculate the texel coordinate that this thread will modify
    const uint32_t index = blockIdx.x*blockDim.x + threadIdx.x;
    const uint32_t sindex = (KERNELRADIUS-1) + index%sharedmemCount;
    texel* sharedTexel = (texel*)&s;

    // Ensure we don't go out of bounds
    if (index >= size)
        return;

    // Calculate the texel coordinates
    float r=0, g=0, b=0, n=0;
    const int idy = index/width;
    const uint32_t uppery = sindex+(KERNELRADIUS-1);

    // Copy shared memory
    sharedTexel[sindex] = img[index];
    if (sindex == (KERNELRADIUS-1))
        for (int i=min(idy, KERNELRADIUS)-1; i>0; i--)
            sharedTexel[sindex-i] = img[index-i];
    if (sindex+1 == sharedmemCount)
        for (int i=min(height-idy, KERNELRADIUS)-1; i>0; i--)
            sharedTexel[sindex+i] = img[index+i];
    __syncthreads();

    // And now a second vertical pass
    for (int y=sindex-(KERNELRADIUS-1), k=-min(0, idy-(KERNELRADIUS-1)); y<uppery; y++)
    {
        const float scale = filter.data[k++];
        const texel* ti = &sharedTexel[y];
        r += scale*ti->r;
        g += scale*ti->g;
        b += scale*ti->b;
        n += scale;
    }
    n = 1/n;

    // Update the output image's desaturated + blurred texel value 
    const color nr = r*n, ng = g*n, nb = b*n;
    const float grey = (1-desaturation)*(0.3*nr + 0.59*ng + 0.11*nb);
    texel* to = &img[index];
    to->r = desaturation*nr + grey;
    to->g = desaturation*ng + grey;
    to->b = desaturation*nb + grey;
}


/*==============================
    gaussian
    Calculates a coordinate on the gaussian matrix
    @param The gaussian matrix X coordinate
    @param The gaussian matrix Y coordinate
==============================*/

float gaussian(const float x, const float y) // I tried to inline this, but apparently GCC on linux doesn't behave the way I expected...
{
    const float sigma = KERNELRADIUS/2;
    const float sigmasqu = sigma*sigma;
    const float mean = KERNELSIZE/2;
    return exp(-0.5*(pow((x-mean)/sigma, 2.0) + pow((y-mean)/sigma, 2.0)))/(TWO_PI*sigmasqu);
}


/*==============================
    main
    Program entrypoint
    @param The number of arguments to the program
    @param A list of strings with the program arguments
==============================*/

int main(int argc, char* argv[]) 
{
    struct hipDeviceProp_t properties;
    uint32_t threadCount, blockCount;
    uint32_t sharedmemSize, sharedmemCount;
    uint32_t imgh, imgw, imgsize;
    uint32_t texelsize;
    texel* img;
    float saturation = 0.5; // default value
    blurMatrix filter;

    // Check if program arguments exist
    if (argc != 2 && argc != 3)
    {
        fprintf(stderr, "usage: %s img.ppm [saturation]\n", argv[0]);
        return EXIT_FAILURE;
    }
    
    // Get the saturation argument if it exists
    if (argc == 3)
        saturation = atof(argv[2]);

    // Open the image file
    FILE *f = fopen(argv[1], "r");
    if (f == NULL) 
    {
        fprintf(stderr, "can't read file %s\n", argv[1]);
        return EXIT_FAILURE;
    }

    // Read the image file
    read_ppm(f, &img, &imgw, &imgh, &texelsize);
    printf("PPM image %ux%u. Texel size = %u\n", imgw, imgh, texelsize);
    imgsize = imgw*imgh;
    #if PRINT_PPM
        printImg(imgw, imgh, img);
    #endif
    fclose(f);

    // Calculate the gaussian blur matrix
    for (uint32_t x=0; x<KERNELSIZE; x++)
        filter.data[x] = gaussian(x, KERNELRADIUS-1);

    // Setup CUDA
    hipGetDeviceProperties(&properties, 0);
    threadCount = fmin(imgsize, properties.maxThreadsPerBlock)/KERNELSIZE;
    blockCount = (int)ceil(((float)(imgsize))/threadCount)*KERNELSIZE;
    sharedmemSize = properties.sharedMemPerBlock;
    sharedmemCount = sharedmemSize/sizeof(texel);
    printf("Creating kernel with %d blocks of %d threads each\n", blockCount, threadCount); 
    printf("Shared memory = %u bytes\n", sharedmemSize);

    // Allocate memory for the outputted image data buffer
    texel *out = (texel*)malloc(imgsize*sizeof(texel));
    assert(out!=NULL);

    // Allocate all the memory for the GPU
    texel *d_in, *d_out;
    hipMalloc(&d_in, imgsize*sizeof(texel));
    hipMalloc(&d_out, imgsize*sizeof(texel));

    // Get the current CPU time
    clock_t t = clock();

    // Copy our image data to the GPU
    hipMemcpy(d_in, img, imgsize*sizeof(texel), hipMemcpyHostToDevice);

    // Get the CPU time after the memory copy
    clock_t ta = clock();

    // Apply the first shader pass (Horizontal blur)
    shader_pass1<<<blockCount, threadCount, sharedmemSize>>>(d_out, d_in, imgsize, imgw, imgh, filter, sharedmemCount-KERNELSIZE*2);
    hipDeviceSynchronize();

    // Apply the second shader pass (Vertical blur + Desaturation)
    shader_pass2<<<blockCount, threadCount, sharedmemSize>>>(d_out, imgsize, imgw, imgh, filter, 1-saturation, sharedmemCount-KERNELSIZE*2);
    hipDeviceSynchronize();

    // Calculate how long the algorithm took
    ta = clock()-ta;

    // Check if the kernel even executed
    hipError_t err=hipGetLastError();
    if (err!=hipSuccess) {
        fprintf(stderr, "err=%u %s\n%s\n", (unsigned) err, hipGetErrorString(err),
                "Problems executing kernel");
        exit(1);
    }

    // Get the calculated value from the GPU
    hipMemcpy(out, d_out, imgsize*sizeof(texel), hipMemcpyDeviceToHost);

    // Calculate and print how long it took to execute 
    t = clock()-t;
    printf("time %f ms (algorithm %f ms)\n", t/(double)(CLOCKS_PER_SEC/1000), ta/(double)(CLOCKS_PER_SEC/1000));

    // Write the output to a new image file
    #if PRINT_PPM
        printImg(imgw, imgh, out);
    #endif
    f = fopen("out.ppm", "w");
    write_ppm(f, out, imgw, imgh, texelsize);
    fclose(f);

    // Exit the program
    return EXIT_SUCCESS;
}