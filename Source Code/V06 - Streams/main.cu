#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <assert.h>
#include <time.h>
#include <ctype.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>



/*********************************
              Macros
*********************************/

// Number of CUDA streams
#define STREAMCOUNT  4

// Print the image before and after to the console
#define PRINT_PPM    0

// The radius of the blur matrix kernel
#define KERNELRADIUS 2

// Useful constants. Don't touch
#define KERNELSIZE   ((2*KERNELRADIUS)-1)
#define TWO_PI 6.28319
#define EULER 2.71828


/*********************************
            Structures
*********************************/

// Texel struct
typedef uint8_t color;
typedef struct {
    color r;
    color g;
    color b;
    color alignment; // Unused, exists for 32 bit memory alignment.
} texel;

// Blur matrix struct (workaround for passing arrays by value)
typedef struct {
  float data[KERNELSIZE];
} blurMatrix;


/*==============================
    read_ppm
    Reads a PPM image ascii file
    @uint32_tparam The pointer to the image file handler 
    @param The pointer to an array of texels
    @param A pointer to store the image width 
    @param A pointer to store the image height
    @param A pointer to store the size of each texel 
==============================*/

void read_ppm(FILE* f, texel** img, uint32_t* width, uint32_t* height, uint32_t* texelsize)
{
    color r, g, b;
    int c;
    uint32_t count = 0;
    char ppm[10];

    // Read the header to get the image properties
    while ((c = fgetc(f)) != EOF && count < 4)
    {

        // Ignore whitespace
        if (isspace(c)) 
            continue;

        // Ignore comments
        if (c == '#') 
        {
            while (fgetc(f) != '\n')
                ;
            continue;
        }

        // Read header elements
        ungetc(c, f);
        switch (count) 
        {
            case 0: count += fscanf(f, "%2s", ppm); break;
            case 1: count += fscanf(f, "%u%u%u", width, height, texelsize); break;
            case 2: count += fscanf(f, "%u%u", height, texelsize); break;
            case 3: count += fscanf(f, "%u", texelsize);
        }
    }

    // Validate what we read
    assert(c != EOF);
    assert(!strcmp("P3", ppm));

    // Allocate memory for the image data
    *img = (texel*)malloc((*width)*(*height)*sizeof(texel));
    assert(img != NULL);

    // Read the texel data from the file
    int pos = 0;
    while (fscanf(f,"%hhu%hhu%hhu", &r, &g, &b) == 3)
    {
        texel* t = &(*img)[pos];
        t->r = r;
        t->g = g;
        t->b = b;
        pos++;
    }
    assert(pos == (*width)*(*height));
}


/*==============================
    write_ppm
    Writes a PPM image ascii file
    @param The pointer to the image file handler 
    @param The image data
    @param The image width (in texels)
    @param The image height (in texels)
    @param The size of each texel 
==============================*/

void write_ppm(FILE* f, texel* img, const uint32_t width, const uint32_t height, const uint32_t texelsize)
{
    // Write the header
    fprintf(f, "P3\n%u %u %u\n", width, height, texelsize);

    // Write the texel data
    for (uint32_t l=0; l<height; l++)
    {
        for (uint32_t c=0; c<width; c++) 
        {
            uint32_t p = (l*width + c);
            fprintf(f, "%hhu %hhu %hhu  ", img[p].r, img[p].g, img[p].b);
        }
        putc('\n', f);
    }
}


/*==============================
    write_ppm
    Prints a PPM image to the screen
    @param The image width (in texels)
    @param The image height (in texels)
    @param The image data
==============================*/

void printImg(const uint32_t imgw, const uint32_t imgh, const texel* img)
{
    for (uint32_t j=0; j<imgh; j++) 
    {
        for (uint32_t i=0; i<imgw; i++) 
        {
            uint32_t x = (i+j*imgw);
            printf("%hhu,%hhu,%hhu  ", img[x].r, img[x].g, img[x].b);
        }
        putchar('\n');
    }
    putchar('\n');
}


/*==============================
    shader_pass1
    Performs a vertical gaussian blur with CUDA
    @param The image array to output to
    @param The image array to read from
    @param The stream offset
    @param The size of the image (in texels)
    @param The image width (in texels)
    @param The image height (in texels)
    @param The gaussian blur matrix
==============================*/

__global__ void shader_pass1(texel* out, texel* in, const uint32_t offset, const uint32_t size, const uint32_t width, const uint32_t height, const blurMatrix filter)
{
    // Calculate the texel coordinate that this thread will modify
    const uint32_t index = offset + blockIdx.x*blockDim.x + threadIdx.x;

    // Ensure we don't go out of bounds
    if (index >= size)
        return;

    // Calculate the texel coordinates
    float r=0, g=0, b=0, n=0;
    const int idx = index%width;
    const int idy = index/width;
    const uint32_t uppery = width*min((idy+(KERNELRADIUS-1)), height);

    // Perform a first vertical gaussian blur pass
    for (int y=max(0, ((int)width)*(idy-(KERNELRADIUS-1))), k=-min(0, idy-(KERNELRADIUS-1)); y<uppery; y+=width)
    {
        const float scale = filter.data[k++];
        const texel* ti = &in[idx+y];
        r += scale*ti->r;
        g += scale*ti->g;
        b += scale*ti->b;
        n += scale;
    }
    n = 1/n;

    // Store this first pass
    texel* to = &out[index];
    to->r = r*n;
    to->g = g*n;
    to->b = b*n;
}


/*==============================
    shader_pass2
    Performs a horizontal gaussian blur, and then desaturation with CUDA
    @param The image array to output to
    @param The stream offset
    @param The size of the image (in texels)
    @param The image width (in texels)
    @param The image height (in texels)
    @param The gaussian blur matrix
    @param The amount of saturation (ranging from 0 to 1).
    @param The amount of desaturation (ranging from 0 to 1).
==============================*/

__global__ void shader_pass2(texel* img, const uint32_t offset, const uint32_t size, const uint32_t width, const uint32_t height, const blurMatrix filter, const float saturation, const float desaturation)
{
    // Calculate the texel coordinate that this thread will modify
    const uint32_t index = offset + blockIdx.x*blockDim.x + threadIdx.x;

    // Ensure we don't go out of bounds
    if (index >= size)
        return;

    // Calculate the texel coordinates
    float r=0, g=0, b=0, n=0;
    const int idx = index%width;
    const int idy = index/width;
    const uint32_t upperx = min(idx+(KERNELRADIUS-1), width);

    // And now a second horizontal pass
    const uint32_t ystart = idy*width;
    for (int x=max(0, idx-(KERNELRADIUS-1)), k=-min(0, idx-(KERNELRADIUS-1)); x<upperx; x++)
    {
        const float scale = filter.data[k++];
        const texel* ti = &img[x+ystart];
        r += scale*ti->r;
        g += scale*ti->g;
        b += scale*ti->b;
        n += scale;
    }
    n = 1/n;

    // Update the output image's desaturated + blurred texel value 
    const color nr = r*n, ng = g*n, nb = b*n;
    const float grey = saturation*(0.3*nr + 0.59*ng + 0.11*nb);
    texel* to = &img[index];
    to->r = desaturation*nr + grey;
    to->g = desaturation*ng + grey;
    to->b = desaturation*nb + grey;
}


/*==============================
    gaussian
    Calculates a coordinate on the gaussian matrix
    @param The gaussian matrix X coordinate
    @param The gaussian matrix Y coordinate
==============================*/

float gaussian(const float x, const float y) // I tried to inline this, but apparently GCC on linux doesn't behave the way I expected...
{
    const float sigma = KERNELRADIUS/2;
    const float sigmasqu = sigma*sigma;
    const float mean = KERNELSIZE/2;
    return exp(-0.5*(pow((x-mean)/sigma, 2.0) + pow((y-mean)/sigma, 2.0)))/(TWO_PI*sigmasqu);
}


/*==============================
    main
    Program entrypoint
    @param The number of arguments to the program
    @param A list of strings with the program arguments
==============================*/

int main(int argc, char* argv[]) 
{
    struct hipDeviceProp_t properties;
    uint32_t streamSize;
    uint32_t threadCount, blockCount, streamCount;
    uint32_t imgh, imgw, imgsize;
    uint32_t texelsize;
    texel* img;
    float saturation = 0.5; // default value
    blurMatrix filter;

    // Check if program arguments exist
    if (argc != 2 && argc != 3)
    {
        fprintf(stderr, "usage: %s img.ppm [saturation]\n", argv[0]);
        return EXIT_FAILURE;
    }
    
    // Get the saturation argument if it exists
    if (argc == 3)
        saturation = atof(argv[2]);

    // Open the image file
    FILE *f = fopen(argv[1], "r");
    if (f == NULL) 
    {
        fprintf(stderr, "can't read file %s\n", argv[1]);
        return EXIT_FAILURE;
    }

    // Read the image file
    read_ppm(f, &img, &imgw, &imgh, &texelsize);
    printf("PPM image %ux%u. Texel size = %u\n", imgw, imgh, texelsize);
    imgsize = imgw*imgh;
    #if PRINT_PPM
        printImg(imgw, imgh, img);
    #endif
    fclose(f);

    // Calculate the gaussian blur matrix
    for (uint32_t x=0; x<KERNELSIZE; x++)
        filter.data[x] = gaussian(x, KERNELRADIUS-1);

    // Setup CUDA
    hipGetDeviceProperties(&properties, 0);
    threadCount = fmin(imgsize, properties.maxThreadsPerBlock);

    // Allocate memory for the outputted image data buffer
    texel *out = (texel*)malloc(imgsize*sizeof(texel));
    assert(out!=NULL);

    // Allocate all the memory for the GPU
    texel *d_in, *d_out;
    hipMalloc(&d_in, imgsize*sizeof(texel));
    hipMalloc(&d_out, imgsize*sizeof(texel));

    // Calculate the number of streams we'll need
    streamCount = STREAMCOUNT;
    streamSize = ceil(((float)imgsize)/streamCount);
    blockCount = ceil(((float)streamSize)/threadCount);
    hipStream_t* streams = (hipStream_t*)malloc(streamCount*sizeof(hipStream_t));
    printf("Creating kernel with %u streams containing %u blocks of %u threads each\n", streamCount, blockCount, threadCount); 

    // Get the current CPU time
    clock_t t = clock();

    // Apply the first shader pass (Vertical blur)
    int lastMemcpyOffset = 0;
    for (int i=0; i<streamCount; i++)
    {
        int offset = i*streamSize;
        int memcpySize = streamSize + imgw*(KERNELRADIUS-1)+(KERNELRADIUS-1);

        // Create the stream and launch the kernel
        hipStreamCreate(&streams[i]);
        hipMemcpyAsync(d_in+lastMemcpyOffset, img+lastMemcpyOffset, fmin(memcpySize, imgsize-lastMemcpyOffset)*sizeof(texel), hipMemcpyHostToDevice, streams[i]);
        shader_pass1<<<blockCount, threadCount, 0, streams[i]>>>(d_out, d_in, offset, imgsize, imgw, imgh, filter);

        // Calculate the memory copy offsets
        lastMemcpyOffset += memcpySize;
    }

    // Apply the second shader pass (Horizontal blur + Desaturation)
    for (int i=0; i<streamCount; i++)
    {
        int offset = i*streamSize;

        // Launch the kernel and download the result
        shader_pass2<<<blockCount, threadCount, 0, streams[i]>>>(d_out, offset, imgsize, imgw, imgh, filter, saturation, 1-saturation);
        hipMemcpyAsync(out+offset, d_out+offset, fmin(streamSize, imgsize-offset)*sizeof(texel), hipMemcpyDeviceToHost, streams[i]);
    }
    hipDeviceSynchronize();

    // Check if the kernel even executed
    hipError_t err=hipGetLastError();
    if (err!=hipSuccess) {
        fprintf(stderr, "err=%u %s\n%s\n", (unsigned) err, hipGetErrorString(err),
                "Problems executing kernel");
        exit(1);
    }

    // Calculate and print how long it took to execute 
    t = clock()-t;
    printf("time %f ms\n", t/(double)(CLOCKS_PER_SEC/1000));

    // Write the output to a new image file
    #if PRINT_PPM
        printImg(imgw, imgh, out);
    #endif
    f = fopen("out.ppm", "w");
    write_ppm(f, out, imgw, imgh, texelsize);
    fclose(f);

    // Exit the program
    return EXIT_SUCCESS;
}