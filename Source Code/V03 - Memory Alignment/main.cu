#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <assert.h>
#include <time.h>
#include <ctype.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>



/*********************************
              Macros
*********************************/

// Print the image before and after to the console
#define PRINT_PPM    0

// The radius of the blur matrix kernel
#define KERNELRADIUS 2

// Useful constants. Don't touch
#define KERNELSIZE   ((2*KERNELRADIUS)-1)
#define TWO_PI 6.28319
#define EULER 2.71828


/*********************************
            Structures
*********************************/

// Texel struct
typedef uint8_t color;
typedef struct {
    color r;
    color g;
    color b;
    color alignment; // Unused, exists for 32 bit memory alignment.
} texel;

// Blur matrix struct (workaround for passing arrays by value)
typedef struct {
  float data[KERNELSIZE][KERNELSIZE];
} blurMatrix;


/*==============================
    read_ppm
    Reads a PPM image ascii file
    @uint32_tparam The pointer to the image file handler 
    @param The pointer to an array of texels
    @param A pointer to store the image width 
    @param A pointer to store the image height
    @param A pointer to store the size of each texel 
==============================*/

void read_ppm(FILE* f, texel** img, uint32_t* width, uint32_t* height, uint32_t* texelsize)
{
    color r, g, b;
    int c;
    uint32_t count = 0;
    char ppm[10];

    // Read the header to get the image properties
    while ((c = fgetc(f)) != EOF && count < 4)
    {

        // Ignore whitespace
        if (isspace(c)) 
            continue;

        // Ignore comments
        if (c == '#') 
        {
            while (fgetc(f) != '\n')
                ;
            continue;
        }

        // Read header elements
        ungetc(c, f);
        switch (count) 
        {
            case 0: count += fscanf(f, "%2s", ppm); break;
            case 1: count += fscanf(f, "%u%u%u", width, height, texelsize); break;
            case 2: count += fscanf(f, "%u%u", height, texelsize); break;
            case 3: count += fscanf(f, "%u", texelsize);
        }
    }

    // Validate what we read
    assert(c != EOF);
    assert(!strcmp("P3", ppm));

    // Allocate memory for the image data
    *img = (texel*)malloc((*width)*(*height)*sizeof(texel));
    assert(img != NULL);

    // Read the texel data from the file
    int pos = 0;
    while (fscanf(f,"%hhu%hhu%hhu", &r, &g, &b) == 3)
    {
        texel* t = &(*img)[pos];
        t->r = r;
        t->g = g;
        t->b = b;
        pos++;
    }
    assert(pos == (*width)*(*height));
}


/*==============================
    write_ppm
    Writes a PPM image ascii file
    @param The pointer to the image file handler 
    @param The image data
    @param The image width (in texels)
    @param The image height (in texels)
    @param The size of each texel 
==============================*/

void write_ppm(FILE* f, texel* img, const uint32_t width, const uint32_t height, const uint32_t texelsize)
{
    // Write the header
    fprintf(f, "P3\n%u %u %u\n", width, height, texelsize);

    // Write the texel data
    for (uint32_t l=0; l<height; l++)
    {
        for (uint32_t c=0; c<width; c++) 
        {
            uint32_t p = (l*width + c);
            fprintf(f, "%hhu %hhu %hhu  ", img[p].r, img[p].g, img[p].b);
        }
        putc('\n', f);
    }
}


/*==============================
    write_ppm
    Prints a PPM image to the screen
    @param The image width (in texels)
    @param The image height (in texels)
    @param The image data
==============================*/

void printImg(const uint32_t imgw, const uint32_t imgh, const texel* img)
{
    for (uint32_t j=0; j<imgh; j++) 
    {
        for (uint32_t i=0; i<imgw; i++) 
        {
            uint32_t x = (i+j*imgw);
            printf("%hhu,%hhu,%hhu  ", img[x].r, img[x].g, img[x].b);
        }
        putchar('\n');
    }
    putchar('\n');
}


/*==============================
    areaFilter
    Performs a gaussian blur with CUDA
    @param The image array to output to
    @param The image array to read from
    @param The size of the image (in texels)
    @param The image width (in texels)
    @param The image height (in texels)
    @param The gaussian blur matrix
==============================*/

__global__ void areaFilter(texel* out, texel* in, const uint32_t size, const uint32_t width, const uint32_t height, const blurMatrix filter)
{
    // Calculate the texel coordinate that this thread will modify
    const uint32_t index = blockIdx.x*blockDim.x + threadIdx.x;

    // Ensure we don't go out of bounds
    if (index >= size)
        return;

    // Calculate the texel coordinates
    float r=0, g=0, b=0, n=0;
    const int idx = index%width;
    const int idy = index/width;
    const uint32_t lowerx = max(0, idx-(KERNELRADIUS-1));
    const uint32_t upperx = min(idx+(KERNELRADIUS-1), width);
    const uint32_t uppery = width*min((idy+(KERNELRADIUS-1)), height);

    // Perform a single pass gaussian blur
    for (int y=max(0, ((int)width)*(idy-(KERNELRADIUS-1))), fy=-min(0, idy-(KERNELRADIUS-1)); y<uppery; y+=width, fy++)
    {
        for (int x=lowerx, fx=-min(0, idx-(KERNELRADIUS-1)); x<upperx; x++, fx++)
        {
            const float scale = filter.data[fx][fy];
            const texel* ti = &in[x+y];
            r += scale*ti->r;
            g += scale*ti->g;
            b += scale*ti->b;
            n += scale;
        }
    }
    n = 1/n;

    // Update the output image's texel value
    texel* to = &out[index];
    to->r = r*n;
    to->g = g*n;
    to->b = b*n;
}


/*==============================
    pointFilter
    Desaturates an image with CUDA
    @param The image array to output to
    @param The image array to read from
    @param The size of the image (in texels)
    @param The amount of saturation (ranging from 0 to 1).
    @param The amount of desaturation (ranging from 0 to 1).
==============================*/

__global__ void pointFilter(texel* out, const uint32_t size, const float saturation, const float desaturation)
{
    // Calculate the texel coordinate that this thread will modify
    const unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;

    // Ensure we don't go out of bounds
    if (index >= size)
        return;

    // Get the color information for this texel
    const texel* ti = &out[index];
    const color r = ti->r;
    const color g = ti->g;
    const color b = ti->b;
    const float grey = saturation*(0.3*r + 0.59*g + 0.11*b);

    // Update the output image's texel value
    texel* to = &out[index];
    to->r = desaturation*r + grey;
    to->g = desaturation*g + grey;
    to->b = desaturation*b + grey;
}


/*==============================
    gaussian
    Calculates a coordinate on the gaussian matrix
    @param The gaussian matrix X coordinate
    @param The gaussian matrix Y coordinate
==============================*/

float gaussian(const float x, const float y) // I tried to inline this, but apparently GCC on linux doesn't behave the way I expected...
{
    const float sigma = KERNELRADIUS/2;
    const float sigmasqu = sigma*sigma;
    const float mean = KERNELSIZE/2;
    return exp(-0.5*(pow((x-mean)/sigma, 2.0) + pow((y-mean)/sigma, 2.0)))/(TWO_PI*sigmasqu);
}


/*==============================
    main
    Program entrypoint
    @param The number of arguments to the program
    @param A list of strings with the program arguments
==============================*/

int main(int argc, char* argv[]) 
{
    struct hipDeviceProp_t properties;
    uint32_t threadCount, blockCount;
    uint32_t imgh, imgw, imgsize;
    uint32_t texelsize;
    texel* img;
    float saturation = 0.5; // default value
    blurMatrix filter;

    // Check if program arguments exist
    if (argc != 2 && argc != 3)
    {
        fprintf(stderr, "usage: %s img.ppm [saturation]\n", argv[0]);
        return EXIT_FAILURE;
    }
    
    // Get the saturation argument if it exists
    if (argc == 3)
        saturation = atof(argv[2]);

    // Open the image file
    FILE *f = fopen(argv[1], "r");
    if (f == NULL) 
    {
        fprintf(stderr, "can't read file %s\n", argv[1]);
        return EXIT_FAILURE;
    }

    // Read the image file
    read_ppm(f, &img, &imgw, &imgh, &texelsize);
    printf("PPM image %ux%u. Texel size = %u\n", imgw, imgh, texelsize);
    imgsize = imgw*imgh;
    #if PRINT_PPM
        printImg(imgw, imgh, img);
    #endif
    fclose(f);

    // Calculate the gaussian blur matrix
    for (uint32_t x=0; x<KERNELSIZE; x++)
        for (uint32_t y=0; y<KERNELSIZE; y++)
            filter.data[x][y] = gaussian(x, y);

    // Setup CUDA
    hipGetDeviceProperties(&properties, 0);
    threadCount = fmin(imgsize, properties.maxThreadsPerBlock);
    blockCount = (int)ceil(((float)(imgsize))/threadCount);
    printf("Creating kernel with %u blocks of %u threads each\n", blockCount, threadCount); 

    // Allocate memory for the outputted image data buffer
    texel *out = (texel*)malloc(imgsize*sizeof(texel));
    assert(out!=NULL);

    // Allocate all the memory for the GPU
    texel *d_in, *d_out;
    hipMalloc(&d_in, imgsize*sizeof(texel));
    hipMalloc(&d_out, imgsize*sizeof(texel));

    // Get the current CPU time
    clock_t t = clock();

    // Copy our image data to the GPU
    hipMemcpy(d_in, img, imgsize*sizeof(texel), hipMemcpyHostToDevice);

    // Get the CPU time after the memory copy
    clock_t ta = clock();

    // Apply a gaussian blur
    areaFilter<<<blockCount, threadCount>>>(d_out, d_in, imgsize, imgw, imgh, filter);
    hipDeviceSynchronize();

    // Apply a grayscale color correction
    pointFilter<<<blockCount, threadCount>>>(d_out, imgsize, saturation, 1-saturation);
    hipDeviceSynchronize();

    // Calculate how long the algorithm took
    ta = clock()-ta;

    // Check if the kernel even executed
    hipError_t err=hipGetLastError();
    if (err!=hipSuccess) {
        fprintf(stderr, "err=%u %s\n%s\n", (unsigned) err, hipGetErrorString(err),
                "Problems executing kernel");
        exit(1);
    }

    // Get the calculated value from the GPU
    hipMemcpy(out, d_out, imgsize*sizeof(texel), hipMemcpyDeviceToHost);

    // Calculate and print how long it took to execute 
    t = clock()-t;
    printf("time %f ms (algorithm %f ms)\n", t/(double)(CLOCKS_PER_SEC/1000), ta/(double)(CLOCKS_PER_SEC/1000));

    // Write the output to a new image file
    #if PRINT_PPM
        printImg(imgw, imgh, out);
    #endif
    f = fopen("out.ppm", "w");
    write_ppm(f, out, imgw, imgh, texelsize);
    fclose(f);

    // Exit the program
    return EXIT_SUCCESS;
}